#include "hip/hip_runtime.h"
#include "CG.cuh"


__global__ void r1_div_x(double *r1, double *r0, double *b) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid == 0) {
        b[0] = r1[0] / r0[0];
    }
}
  
__global__ void a_minus(double *a, double *na) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid == 0) {
        na[0] = -(a[0]);
    }
}

__device__ void gpuSpMV(int *row, int *col, double *val, int nz, int N, double alpha, double *x_vec, double *Ax, cg::thread_block &cta, const cg::grid_group &grid)
{
    for (int i = grid.thread_rank(); i < N; i += grid.size()) {
        int row_elem = row[i];
        int next_row_elem = row[i + 1];
        int num_elems_this_row = next_row_elem - row_elem;

        double output = 0.0;
        for (int j = 0; j < num_elems_this_row; j++) {
            output += alpha * val[row_elem + j] * x_vec[col[row_elem + j]];
        }
        Ax[i] = output;
    }
}

__device__ void gpuDaxpy(double *x, double *y, double a, int size, const cg::grid_group &grid)
{
    for (int i = grid.thread_rank(); i < size; i += grid.size()) {
        y[i] += a * x[i] + y[i];
    }
}

__device__ void gpuDcopy(double *x, double *y, int size, const cg::grid_group &grid)
{
    for (int i = grid.thread_rank(); i < size; i += grid.size()) {
        y[i] = x[i];
    }
}

__device__ void gpuDaxpby(const double *x, double *y, double a, double b, int size, const cg::grid_group &grid)
{
    for (int i = grid.thread_rank(); i < size; i += grid.size()) {
        y[i] = a * x[i] + b * y[i];
    }
}

__device__ void gpuDdot(double *x, double *y, double *result, int size, const cg::thread_block &cta, const cg::grid_group &grid)
{
    extern __shared__ double tmp[];

    double temp_sum = 0.0;
    for (int i = grid.thread_rank(); i < size; i += grid.size()) {
        temp_sum += static_cast<double>(x[i] * y[i]);
    }

    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

    temp_sum = cg::reduce(tile32, temp_sum, cg::plus<double>());

    if (tile32.thread_rank() == 0) {
        tmp[tile32.meta_group_rank()] = temp_sum;    
    }

    cg::sync(cta);

    if (tile32.meta_group_rank() == 0) {
        temp_sum = tile32.thread_rank() < tile32.meta_group_size() ? tmp[tile32.thread_rank()] : 0.0;
        temp_sum = cg::reduce(tile32, temp_sum, cg::plus<double>());

        if (tile32.thread_rank() == 0) {
            atomicAdd(result, temp_sum);
        }
    }
}

extern "C" __global void CG_MBCG_kernel(int *row, int *col, double *val, double *x_vec, double *Ax, double *p, double *r, double *dot_result, int nz, int N, double tolerance, int limit)
{
    cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();

    double alpha = 1.0;
    double alpham1 = -1.0;
    double r0 = 0.0, r1, bb, a, na;

    gpuSpMV(row, col, val, nz, N, alpha, x_vec, Ax, cta, grid);
    cg::sync(grid);

    gpuDaxpy(Ax, r, alpham1, N, grid);
    cg::sync(grid);

    gpuDdot(r, r, dot_result, N, cta, grid);
    cg::sync(grid);

    r1 = *dot_result;

    int k = 1;
    while (r1 > tolerance * tolerance && k <= limit) {
        if (k > 1) {
            bb = r1 / r0;
            gpuDaxpby(r, p, alpha, bb, N, grid);
        } else {
            gpuDcopy(r, p, N, grid);
        }

        cg::sync(grid);

        gpuSpMV(row, col, val, nz, N, alpha, p, Ax, cta, grid);

        if (threadIdx.x == 0 && blockIdx.x == 0) {
            *dot_result = 0.0;
        }

        cg::sync(grid);

        gpuDdot(p, Ax, dot_result, N, cta, grid);

        cg::sync(grid);

        a = r1 / *dot_result;

        gpuDaxpy(p, x_vec, a, N, grid);
        na = -a;
        gpuDaxpy(Ax, r, na, N, grid);

        r0 = r1;

        cg::sync(grid);
        if (threadIdx.x == 0 && blockIdx.x == 0) {
            *dot_result = 0.0;
        }

        cg::sync(grid);

        gpuDdot(r, r, dot_result, N, cta, grid);

        cg::sync(grid);

        r1 = *dot_result;
        k++;
    }

}

void CG_MBCG(const SymetrixSparseMatrix& A,Vector& x,const Vector& b,double tolerance,int limit,int& iter,double& norm)
{
    printf("CG with Multi_Block Cooperative_Groups...\n");
    auto m_Mat = A.getMat();
    int num_rows = m_Mat.size(), nz = 0;
    int N = num_rows;
    int num_offsets = N + 1;

    int *row, *col;
    CUDACheck(hipMallocManaged(reinterpret_cast<void **>(&row), num_offsets * sizeof(int)));
    row[0] = 0;
    for (int i = 0; i < m_Mat.size(); i++) {
		for (int j = 0; j < m_Mat[i].size(); j++) {
			if (m_Mat[i][j].second != 0.f) {
				nz++;
			}
		}
        row[i + 1] = nz;
	}
	
    CUDACheck(hipMallocManaged(reinterpret_cast<void **>(&col), nz * sizeof(int)));
    double *val, *x_vec, *rhs;
    CUDACheck(hipMallocManaged(reinterpret_cast<void **>(&val), nz * sizeof(double)));
    CUDACheck(hipMallocManaged(reinterpret_cast<void **>(&x_vec), N * sizeof(double)));
    CUDACheck(hipMallocManaged(reinterpret_cast<void **>(&rhs), N * sizeof(double)));
    // from ellpack to csr
    int cnt = 0;
    for (int i = 0; i < m_Mat.size(); i++) {
		for (int j = 0; j < m_Mat[i].size(); j++) {
			if (m_Mat[i][j].second != 0.f) {
				h_col[cnt] = m_Mat[i][j].first;
                h_val[cnt] = m_Mat[i][j].second;
                cnt++;
			}
		}
	}
    auto b_vec = b.generateScalar();
    for (int i = 0; i < N; i++) {
        rhs[i] = b_vec[i];
        x_vec[i] = 0.0;
    }
    //--------------------------------------------------------------------------
    double *r, *p, *Ax;
    int k;
    double r1;
    double *dot_result;
    hipEvent_t start, stop;
    
    CUDACheck(hipMallocManaged(reinterpret_cast<void **>(&dot_result), sizeof(double)));
    *dot_result = 0.0;

    CUDACheck(hipMallocManaged(reinterpret_cast<void **>(&r), N * sizeof(double)));
    CUDACheck(hipMallocManaged(reinterpret_cast<void **>(&p), N * sizeof(double)));
    CUDACheck(hipMallocManaged(reinterpret_cast<void **>(&Ax), N * sizeof(double)));
    
    hipDeviceSynchronize();

    CUDACheck(hipEventCreate(&start));
    CUDACheck(hipEventCreate(&stop));
    
    hipDeviceSynchronize();

    for (int i = 0; i < N; i++) {
        r[i] = rhs[i];
    }

    void *kernelArgs[] = {
        (void *)&row, (void *)&col, (void *)&val, (void *)&x_vec, 
        (void *)&Ax, (void *)&p, (void *)&r, (void *)&dot_result,
        (void *)&nz, (void *)&N, (void *)&tolerance, (void *)&limit
    };
    
    int sMemSize = sizeof(double) * ((THREADS_PER_BLOCK / 32) + 1);
    int numBlocksPerSm = 0;
    int numThreads = THREADS_PER_BLOCK;

    CUDACheck(hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &numBlocksPerSm, CG_MBCG_kernel, numThreads, sMemSize));

    int numSms = 32;
    dim3 dimGrid(numSms * numBlocksPerSm, 1, 1), dimBlock(numThreads, 1, 1);
    
    CUDACheck(hipEventRecord(start));
    CUDACheck(hipLaunchCooperativeKernel((void *)CG_MBCG_kernel, dimGrid, dimBlock, kernelArgs, sMemSize, NULL));
    CUDACheck(hipEventRecord(stop, 0));
    hipDeviceSynchronize();

    float time;
    CUDACheck(hipEventElapsedTime(&time, start, stop));

    r1 = *dot_result;
    printf("Final residual = %e, kernel execution time = %f ms\n", std::sqrt(r1), time);
     
    std::vector<double> xx(N);
    for (int i = 0; i < N; i++) {
        xx[i] = x_vec[i];
    }

    // save iteration info
    iter = k;
    norm = std::sqrt(r1);
    x.setvalues({xx.begin(), xx.end()});

    
    CUDACheck(hipFree(row));
    CUDACheck(hipFree(col));
    CUDACheck(hipFree(val));
    CUDACheck(hipFree(x_vec));
    CUDACheck(hipFree(rhs));
    CUDACheck(hipFree(r));
    CUDACheck(hipFree(p));
    CUDACheck(hipFree(Ax));
    CUDACheck(hipFree(dot_result));
    CUDACheck(hipEventDestroy(start));
    CUDACheck(hipEventDestroy(stop));

    return ;
}


void CG_CG(const SymetrixSparseMatrix& A,Vector& x,const Vector& b,double tolerance,int limit,int& iter,double& norm)
{
    printf("CG with CUDA Graph...\n");
    auto m_Mat = A.getMat();
	int num_rows = m_Mat.size(), nz = 0;
	int N = num_rows;
	int num_offsets = N + 1;
    double r1;

    int     *h_row, *h_col;
    double  *h_val, *h_x;
    CUDACheck(hipHostMalloc(&h_row, num_offsets * sizeof(int)));
    h_row[0] = 0;
	for (int i = 0; i < m_Mat.size(); i++) {
		for (int j = 0; j < m_Mat[i].size(); j++) {
			if (m_Mat[i][j].second != 0.f) {
				nz++;
			}
		}
        h_row[i + 1] = nz;
	}
	
    CUDACheck(hipHostMalloc(&h_col, nz * sizeof(int)));
    CUDACheck(hipHostMalloc(&h_val, nz * sizeof(double)));
    CUDACheck(hipHostMalloc(&h_x, N * sizeof(double)));
    double* rhs   = (double*) malloc(N * sizeof(double));
	// from ellpack to csr
    int cnt = 0;
    for (int i = 0; i < m_Mat.size(); i++) {
		for (int j = 0; j < m_Mat[i].size(); j++) {
			if (m_Mat[i][j].second != 0.f) {
				h_col[cnt] = m_Mat[i][j].first;
                h_val[cnt] = m_Mat[i][j].second;
                cnt++;
			}
		}
	}
    auto b_vec = b.generateScalar();
    for (int i = 0; i < N; i++) {
        rhs[i] = b_vec[i];
        h_x[i] = 0.0;
    }
    //--------------------------------------------------------------------------
    int *d_col, *d_row;
    double *d_val, *d_x;
    double *d_r, *d_p, *d_Ax;
    int k;
    double alpha, beta, alpham1;
    
    hipStream_t stream1, streamForGraph;
    
    /* Get handle to the CUBLAS context */
    hipblasHandle_t cublasHandle = 0;
    CUBLASCheck(hipblasCreate(&cublasHandle));
    
    /* Get handle to the CUSPARSE context */
    hipsparseHandle_t cusparseHandle = 0;
    CUSPARSECheck(hipsparseCreate(&cusparseHandle));
    
    CUDACheck(hipStreamCreate(&stream1));
    
    CUDACheck(hipMalloc((void **)&d_col, nz * sizeof(int)));
    CUDACheck(hipMalloc((void **)&d_row, num_offsets * sizeof(int)));
    CUDACheck(hipMalloc((void **)&d_val, nz * sizeof(double)));
    CUDACheck(hipMalloc((void **)&d_x, N * sizeof(double)));
    CUDACheck(hipMalloc((void **)&d_r, N * sizeof(double)));
    CUDACheck(hipMalloc((void **)&d_p, N * sizeof(double)));
    CUDACheck(hipMalloc((void **)&d_Ax, N * sizeof(double)));
    
    double *d_r1, *d_r0, *d_dot, *d_a, *d_na, *d_b;
    CUDACheck(hipMalloc((void **)&d_r1, sizeof(double)));
    CUDACheck(hipMalloc((void **)&d_r0, sizeof(double)));
    CUDACheck(hipMalloc((void **)&d_dot, sizeof(double)));
    CUDACheck(hipMalloc((void **)&d_a, sizeof(double)));
    CUDACheck(hipMalloc((void **)&d_na, sizeof(double)));
    CUDACheck(hipMalloc((void **)&d_b, sizeof(double)));
    
    /* Wrap raw data into cuSPARSE generic API objects */
    hipsparseSpMatDescr_t matA = NULL;
    CUSPARSECheck(hipsparseCreateCsr(&matA, N, N, nz, d_row, d_col, d_val,
                                        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));
    hipsparseDnVecDescr_t vecx = NULL;
    CUSPARSECheck(hipsparseCreateDnVec(&vecx, N, d_x, HIP_R_64F));
    hipsparseDnVecDescr_t vecp = NULL;
    CUSPARSECheck(hipsparseCreateDnVec(&vecp, N, d_p, HIP_R_64F));
    hipsparseDnVecDescr_t vecAx = NULL;
    CUSPARSECheck(hipsparseCreateDnVec(&vecAx, N, d_Ax, HIP_R_64F));
    
    /* Allocate workspace for cuSPARSE */
    size_t bufferSize = 0;
    CUSPARSECheck(hipsparseSpMV_bufferSize(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecx,
        &beta, vecAx, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));
    void *buffer = NULL;
    CUDACheck(hipMalloc(&buffer, bufferSize));
    
    hipsparseMatDescr_t descr = 0;
    CUSPARSECheck(hipsparseCreateMatDescr(&descr));
    
    CUSPARSECheck(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CUSPARSECheck(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));
    
    // int numBlocks = 0, blockSize = 0;
    // CUDACheck(
    //     hipOccupancyMaxPotentialBlockSize(&numBlocks, &blockSize, initVectors));
    
    CUDACheck(hipMemcpyAsync(d_col, h_col, nz * sizeof(int),
                                    hipMemcpyHostToDevice, stream1));
    CUDACheck(hipMemcpyAsync(d_row, h_row, num_offsets * sizeof(int),
                                    hipMemcpyHostToDevice, stream1));
    CUDACheck(hipMemcpyAsync(d_val, h_val, nz * sizeof(double),
                                    hipMemcpyHostToDevice, stream1));
    // r0 = b - A * x
    // initVectors<<<numBlocks, blockSize, 0, stream1>>>(d_r, d_x, N);
    CUDACheck(hipMemcpyAsync(d_r, rhs, N * sizeof(double),
                                    hipMemcpyHostToDevice, stream1));
    CUDACheck(hipMemsetAsync(d_x, 0, N * sizeof(double), stream1));

    alpha = 1.0;
    alpham1 = -1.0;
    beta = 0.0;
    
    CUSPARSECheck(hipsparseSetStream(cusparseHandle, stream1));
    CUSPARSECheck(hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha, matA, vecx, &beta, vecAx, HIP_R_64F,
                                    HIPSPARSE_SPMV_ALG_DEFAULT, buffer));
    
    CUBLASCheck(hipblasSetStream(cublasHandle, stream1));
    CUBLASCheck(hipblasDaxpy(cublasHandle, N, &alpham1, d_Ax, 1, d_r, 1));
    
    CUBLASCheck(
        hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_DEVICE));
    CUBLASCheck(hipblasDdot(cublasHandle, N, d_r, 1, d_r, 1, d_r1));
    
    k = 1;
    // First Iteration when k=1 starts
    CUBLASCheck(hipblasDcopy(cublasHandle, N, d_r, 1, d_p, 1));
    CUSPARSECheck(hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha, matA, vecp, &beta, vecAx, HIP_R_64F,
                                    HIPSPARSE_SPMV_ALG_DEFAULT, buffer));
    
    CUBLASCheck(hipblasDdot(cublasHandle, N, d_p, 1, d_Ax, 1, d_dot));
    
    r1_div_x<<<1, 1, 0, stream1>>>(d_r1, d_dot, d_a);
    
    CUBLASCheck(hipblasDaxpy(cublasHandle, N, d_a, d_p, 1, d_x, 1));
    
    a_minus<<<1, 1, 0, stream1>>>(d_a, d_na);
    
    CUBLASCheck(hipblasDaxpy(cublasHandle, N, d_na, d_Ax, 1, d_r, 1));
    
    CUDACheck(hipMemcpyAsync(d_r0, d_r1, sizeof(double),
                                    hipMemcpyDeviceToDevice, stream1));
    
    CUBLASCheck(hipblasDdot(cublasHandle, N, d_r, 1, d_r, 1, d_r1));
    
    CUDACheck(hipMemcpyAsync(&r1, d_r1, sizeof(double),
                                    hipMemcpyDeviceToHost, stream1));
    CUDACheck(hipStreamSynchronize(stream1));
    printf("iteration = %5d, residual = %e\n", k, std::sqrt(r1));
    // First Iteration when k=1 ends
    k++;
    
#if WITH_GRAPH
    hipGraph_t initGraph;
    CUDACheck(hipStreamCreate(&streamForGraph));
    CUBLASCheck(hipblasSetStream(cublasHandle, stream1));
    CUSPARSECheck(hipsparseSetStream(cusparseHandle, stream1));
    CUDACheck(hipStreamBeginCapture(stream1, hipStreamCaptureModeGlobal));
    
    r1_div_x<<<1, 1, 0, stream1>>>(d_r1, d_r0, d_b);
    hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_DEVICE);
    CUBLASCheck(hipblasDscal(cublasHandle, N, d_b, d_p, 1));
    hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_HOST);
    CUBLASCheck(hipblasDaxpy(cublasHandle, N, &alpha, d_r, 1, d_p, 1));
    hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_DEVICE);
    
    CUSPARSECheck(
        hipsparseSetPointerMode(cusparseHandle, HIPSPARSE_POINTER_MODE_HOST));
    CUSPARSECheck(hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha, matA, vecp, &beta, vecAx, HIP_R_64F,
                                    HIPSPARSE_SPMV_ALG_DEFAULT, buffer));
    
    CUDACheck(hipMemsetAsync(d_dot, 0, sizeof(double), stream1));
    CUBLASCheck(hipblasDdot(cublasHandle, N, d_p, 1, d_Ax, 1, d_dot));
    
    r1_div_x<<<1, 1, 0, stream1>>>(d_r1, d_dot, d_a);
    
    CUBLASCheck(hipblasDaxpy(cublasHandle, N, d_a, d_p, 1, d_x, 1));
    
    a_minus<<<1, 1, 0, stream1>>>(d_a, d_na);
    
    CUBLASCheck(hipblasDaxpy(cublasHandle, N, d_na, d_Ax, 1, d_r, 1));
    
    CUDACheck(hipMemcpyAsync(d_r0, d_r1, sizeof(double),
                                    hipMemcpyDeviceToDevice, stream1));
    CUDACheck(hipMemsetAsync(d_r1, 0, sizeof(double), stream1));
    
    CUBLASCheck(hipblasDdot(cublasHandle, N, d_r, 1, d_r, 1, d_r1));
    
    CUDACheck(hipMemcpyAsync((double *)&r1, d_r1, sizeof(double),
                                    hipMemcpyDeviceToHost, stream1));
    
    CUDACheck(hipStreamEndCapture(stream1, &initGraph));
    hipGraphExec_t graphExec;
    CUDACheck(hipGraphInstantiate(&graphExec, initGraph, NULL, NULL, 0));
#endif
    
    CUBLASCheck(hipblasSetStream(cublasHandle, stream1));
    CUSPARSECheck(hipsparseSetStream(cusparseHandle, stream1));
    
    // iteration
    // r1 > tolerance * tolerance * r0
    while (r1 > tolerance * tolerance && k <= limit) {
#if WITH_GRAPH
        CUDACheck(hipGraphLaunch(graphExec, streamForGraph));
        CUDACheck(hipStreamSynchronize(streamForGraph));
#else
        r1_div_x<<<1, 1, 0, stream1>>>(d_r1, d_r0, d_b);
        hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_DEVICE);
        CUBLASCheck(hipblasDscal(cublasHandle, N, d_b, d_p, 1));
    
        hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_HOST);
        CUBLASCheck(hipblasDaxpy(cublasHandle, N, &alpha, d_r, 1, d_p, 1));
    
        CUSPARSECheck(hipsparseSpMV(
            cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecp,
            &beta, vecAx, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, buffer));
    
        hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_DEVICE);
        CUBLASCheck(hipblasDdot(cublasHandle, N, d_p, 1, d_Ax, 1, d_dot));
    
        r1_div_x<<<1, 1, 0, stream1>>>(d_r1, d_dot, d_a);
    
        CUBLASCheck(hipblasDaxpy(cublasHandle, N, d_a, d_p, 1, d_x, 1));
    
        a_minus<<<1, 1, 0, stream1>>>(d_a, d_na);
        CUBLASCheck(hipblasDaxpy(cublasHandle, N, d_na, d_Ax, 1, d_r, 1));
    
        CUDACheck(hipMemcpyAsync(d_r0, d_r1, sizeof(double),
                                        hipMemcpyDeviceToDevice, stream1));
    
        CUBLASCheck(hipblasDdot(cublasHandle, N, d_r, 1, d_r, 1, d_r1));
        CUDACheck(hipMemcpyAsync((double *)&r1, d_r1, sizeof(double),
                                        hipMemcpyDeviceToHost, stream1));
        CUDACheck(hipStreamSynchronize(stream1));
#endif
        // printf("iteration = %5d, residual = %e\n", k, std::sqrt(r1));
        k++;
    }
    
#if WITH_GRAPH
    CUDACheck(hipMemcpyAsync(h_x, d_x, N * sizeof(double),
                                    hipMemcpyDeviceToHost, streamForGraph));
    CUDACheck(hipStreamSynchronize(streamForGraph));
#else
    CUDACheck(hipMemcpyAsync(h_x, d_x, N * sizeof(double),
                                    hipMemcpyDeviceToHost, stream1));
    CUDACheck(hipStreamSynchronize(stream1));
#endif
    
    // double rsum, diff, err = 0.0;
    
    // for (int i = 0; i < N; i++) {
    //     rsum = 0.0;
    
    //     for (int j = h_row[i]; j < h_row[i + 1]; j++) {
    //         rsum += h_val[j] * h_x[h_col[j]];
    //     }
    
    //     diff = fabs(rsum - rhs[i]);
    
    //     if (diff > err) {
    //         err = diff;
    //     }
    // }
    
    std::vector<double> xx(N);
    for (int i = 0; i < N; i++) {
        xx[i] = h_x[i];
    }

    // save iteration info
    iter = k;
    norm = std::sqrt(r1);
    x.setvalues({xx.begin(), xx.end()});

#if WITH_GRAPH
    CUDACheck(hipGraphExecDestroy(graphExec));
    CUDACheck(hipGraphDestroy(initGraph));
    CUDACheck(hipStreamDestroy(streamForGraph));
#endif
    CUDACheck(hipStreamDestroy(stream1));
    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);
    
    if (matA) {
        CUSPARSECheck(hipsparseDestroySpMat(matA));
    }
    if (vecx) {
        CUSPARSECheck(hipsparseDestroyDnVec(vecx));
    }
    if (vecAx) {
        CUSPARSECheck(hipsparseDestroyDnVec(vecAx));
    }
    if (vecp) {
        CUSPARSECheck(hipsparseDestroyDnVec(vecp));
    }
    
    CUDACheck(hipHostFree(h_row));
    CUDACheck(hipHostFree(h_col));
    CUDACheck(hipHostFree(h_val));
    CUDACheck(hipHostFree(h_x));
    free(rhs);
    CUDACheck(hipFree(d_col));
    CUDACheck(hipFree(d_row));
    CUDACheck(hipFree(d_val));
    CUDACheck(hipFree(d_x));
    CUDACheck(hipFree(d_r));
    CUDACheck(hipFree(d_p));
    CUDACheck(hipFree(d_Ax));
    
    // printf("Test Summary:  Error amount = %f\n", err);
    // exit((k <= limit) ? 0 : 1);
    return ;
}

void CG_UM(const SymetrixSparseMatrix& A,Vector& x,const Vector& b,double tolerance,int limit,int& iter,double& norm)
{
    printf("CG with Unified Memory...\n");
    auto m_Mat = A.getMat();
	int num_rows = m_Mat.size(), nz = 0;
	int N = num_rows;
	int num_offsets = N + 1;

    int *row, *col;
    CUDACheck(hipMallocManaged(&row, num_offsets * sizeof(int)));
    row[0] = 0;
    for (int i = 0; i < m_Mat.size(); i++) {
		for (int j = 0; j < m_Mat[i].size(); j++) {
			if (m_Mat[i][j].second != 0.f) {
				nz++;
			}
		}
        row[i + 1] = nz;
	}
	 
    CUDACheck(hipMallocManaged(&col, nz * sizeof(int)));
    double *val, *x_vec, *rhs;
    CUDACheck(hipMallocManaged(&val, nz * sizeof(double)));
    CUDACheck(hipMallocManaged(&x_vec, N * sizeof(double)));
    CUDACheck(hipMallocManaged(&rhs, N * sizeof(double)));
    // from ellpack to csr
    int cnt = 0;
    for (int i = 0; i < m_Mat.size(); i++) {
		for (int j = 0; j < m_Mat[i].size(); j++) {
			if (m_Mat[i][j].second != 0.f) {
				h_col[cnt] = m_Mat[i][j].first;
                h_val[cnt] = m_Mat[i][j].second;
                cnt++;
			}
		}
	}
    auto b_vec = b.generateScalar();
    for (int i = 0; i < N; i++) {
        rhs[i] = b_vec[i];
        x_vec[i] = 0.0;
    }
    //--------------------------------------------------------------------------
    double *r, *p, *Ax;
    int k;
    double a, bb, na, r0, r1, dot;
    double alpha, beta, alpham1;
    
    /* Get handle to the CUBLAS context */
    hipblasHandle_t cublasHandle = 0;
    CUBLASCheck(hipblasCreate(&cublasHandle));
    
    /* Get handle to the CUSPARSE context */
    hipsparseHandle_t cusparseHandle = 0;
    CUSPARSECheck(hipsparseCreate(&cusparseHandle));
    
    CUDACheck(hipMallocManaged(&r, N * sizeof(double)));
    CUDACheck(hipMallocManaged(&p, N * sizeof(double)));
    CUDACheck(hipMallocManaged(&Ax, N * sizeof(double)));
    
    /* Wrap raw data into cuSPARSE generic API objects */
    hipsparseSpMatDescr_t matA = NULL;
    CUSPARSECheck(hipsparseCreateCsr(&matA, N, N, nz, d_row, d_col, d_val,
                                        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));
    hipsparseDnVecDescr_t vecx = NULL;
    CUSPARSECheck(hipsparseCreateDnVec(&vecx, N, d_x, HIP_R_64F));
    hipsparseDnVecDescr_t vecp = NULL;
    CUSPARSECheck(hipsparseCreateDnVec(&vecp, N, d_p, HIP_R_64F));
    hipsparseDnVecDescr_t vecAx = NULL;
    CUSPARSECheck(hipsparseCreateDnVec(&vecAx, N, d_Ax, HIP_R_64F));
    
    hipDeviceSynchronize();

    for (int i = 0; i < N; i++) {
        r[i] = rhs[i];
    }

    alpha = 1.0;
    alpham1 = -1.0;
    beta = 0.0;
    r0 = 0.0;
    /* Allocate workspace for cuSPARSE */
    size_t bufferSize = 0;
    CUSPARSECheck(hipsparseSpMV_bufferSize(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecx,
        &beta, vecAx, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));
    void *buffer = NULL; 
    CUDACheck(hipMalloc(&buffer, bufferSize));
    
    CUSPARSECheck(hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha, matA, vecx, &beta, vecAx, HIP_R_64F,
                                    HIPSPARSE_SPMV_ALG_DEFAULT, buffer));
    CUBLASCheck(hipblasDaxpy(cublasHandle, N, &alpham1, Ax, 1, r, 1));
    CUBLASCheck(hipblasDdot(cublasHandle, N, r, 1, r, 1, &r1));
    
    k = 1;
    while (r1 > tolerance * tolerance && k <= limit) {
        if (k > 1) {
            bb = r1 / r0;
            CUBLASCheck(hipblasDscal(cublasHandle, N, &bb, p, 1));
            CUBLASCheck(hipblasDaxpy(cublasHandle, N, &alpha, r, 1, p, 1));
        } else {
            CUBLASCheck(hipblasDcopy(cublasHandle, N, r, 1, p, 1));
        }

        CUSPARSECheck(hipsparseSpMV(cusparseHandle, 
            HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecp, &beta, vecAx, 
            HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, buffer));
        CUBLASCheck(hipblasDdot(cublasHandle, N, p, 1, Ax, 1, &dot));
        a = r1 / dot;

        CUBLASCheck(hipblasDaxpy(cublasHandle, N, &a, p, 1, x_vec, 1));
        na = -a;
        CUBLASCheck(hipblasDaxpy(cublasHandle, N, &na, Ax, 1, r, 1));

        r0 = r1;
        CUBLASCheck(hipblasDdot(cublasHandle, N, r, 1, r, 1, &r1));
        hipDeviceSynchronize();
        
        k++;
    }
     
    std::vector<double> xx(N);
    for (int i = 0; i < N; i++) {
        xx[i] = x_vec[i];
    }

    // save iteration info
    iter = k;
    norm = std::sqrt(r1);
    x.setvalues({xx.begin(), xx.end()});

    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);
    
    if (matA) {
        CUSPARSECheck(hipsparseDestroySpMat(matA));
    }
    if (vecx) {
        CUSPARSECheck(hipsparseDestroyDnVec(vecx));
    }
    if (vecAx) {
        CUSPARSECheck(hipsparseDestroyDnVec(vecAx));
    }
    if (vecp) {
        CUSPARSECheck(hipsparseDestroyDnVec(vecp));
    }
    
    CUDACheck(hipFree(row));
    CUDACheck(hipFree(col));
    CUDACheck(hipFree(val));
    CUDACheck(hipFree(x_vec));
    CUDACheck(hipFree(rhs));
    CUDACheck(hipFree(r));
    CUDACheck(hipFree(p));
    CUDACheck(hipFree(Ax));

    return ;
}

void CG(const SymetrixSparseMatrix& A,Vector& x,const Vector& b,double tolerance,int limit,int& iter,double& norm)
{
    printf("CG...\n");
    auto m_Mat = A.getMat();
	int num_rows = m_Mat.size(), nz = 0;
	int N = num_rows;
	int num_offsets = N + 1;

    int *h_row = (int *)malloc(num_offsets * sizeof(int));
    h_row[0] = 0;
	for (int i = 0; i < m_Mat.size(); i++) {
		for (int j = 0; j < m_Mat[i].size(); j++) {
			if (m_Mat[i][j].second != 0.f) {
				nz++;
			}
		}
        h_row[i + 1] = nz;
	}
	 
    int *h_col = (int *)malloc(nz * sizeof(int));
    double *h_val = (double *)malloc(nz * sizeof(double));
    double *h_x = (double *)malloc(N * sizeof(double));
    double *rhs   = (double*) malloc(N * sizeof(double));
	// from ellpack to csr
    int cnt = 0;
    for (int i = 0; i < m_Mat.size(); i++) {
		for (int j = 0; j < m_Mat[i].size(); j++) {
			if (m_Mat[i][j].second != 0.f) {
				h_col[cnt] = m_Mat[i][j].first;
                h_val[cnt] = m_Mat[i][j].second;
                cnt++;
			}
		}
	}
    auto b_vec = b.generateScalar();
    for (int i = 0; i < N; i++) {
        rhs[i] = b_vec[i];
        h_x[i] = 0.0;
    }
    //--------------------------------------------------------------------------
    int *d_col, *d_row;
    double *d_val, *d_x;
    double *d_r, *d_p, *d_Ax;
    int k;
    double a, bb, na, r0, r1, dot;
    double alpha, beta, alpham1;
    
    /* Get handle to the CUBLAS context */
    hipblasHandle_t cublasHandle = 0;
    CUBLASCheck(hipblasCreate(&cublasHandle));
    
    /* Get handle to the CUSPARSE context */
    hipsparseHandle_t cusparseHandle = 0;
    CUSPARSECheck(hipsparseCreate(&cusparseHandle));
    
    CUDACheck(hipMalloc((void **)&d_col, nz * sizeof(int)));
    CUDACheck(hipMalloc((void **)&d_row, num_offsets * sizeof(int)));
    CUDACheck(hipMalloc((void **)&d_val, nz * sizeof(double)));
    CUDACheck(hipMalloc((void **)&d_x, N * sizeof(double)));
    CUDACheck(hipMalloc((void **)&d_r, N * sizeof(double)));
    CUDACheck(hipMalloc((void **)&d_p, N * sizeof(double)));
    CUDACheck(hipMalloc((void **)&d_Ax, N * sizeof(double)));
    
    /* Wrap raw data into cuSPARSE generic API objects */
    hipsparseSpMatDescr_t matA = NULL;
    CUSPARSECheck(hipsparseCreateCsr(&matA, N, N, nz, d_row, d_col, d_val,
                                        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));
    hipsparseDnVecDescr_t vecx = NULL;
    CUSPARSECheck(hipsparseCreateDnVec(&vecx, N, d_x, HIP_R_64F));
    hipsparseDnVecDescr_t vecp = NULL;
    CUSPARSECheck(hipsparseCreateDnVec(&vecp, N, d_p, HIP_R_64F));
    hipsparseDnVecDescr_t vecAx = NULL;
    CUSPARSECheck(hipsparseCreateDnVec(&vecAx, N, d_Ax, HIP_R_64F));
    
    /* Initialize problem data */
    CUDACheck(hipMemcpy(d_col, h_col, nz * sizeof(int), hipMemcpyHostToDevice));
    CUDACheck(hipMemcpy(d_row, h_row, num_offsets * sizeof(int), hipMemcpyHostToDevice));
    CUDACheck(hipMemcpy(d_val, h_val, nz * sizeof(double), hipMemcpyHostToDevice));
    CUDACheck(hipMemcpy(d_x, h_x, N * sizeof(double), hipMemcpyHostToDevice));
    CUDACheck(hipMemcpy(d_r, rhs, N * sizeof(double), hipMemcpyHostToDevice));

    alpha = 1.0;
    alpham1 = -1.0;
    beta = 0.0;
    r0 = 0.0;
    /* Allocate workspace for cuSPARSE */
    size_t bufferSize = 0;
    CUSPARSECheck(hipsparseSpMV_bufferSize(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecx,
        &beta, vecAx, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));
    void *buffer = NULL; 
    CUDACheck(hipMalloc(&buffer, bufferSize));
    
    CUSPARSECheck(hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha, matA, vecx, &beta, vecAx, HIP_R_64F,
                                    HIPSPARSE_SPMV_ALG_DEFAULT, buffer));
    CUBLASCheck(hipblasDaxpy(cublasHandle, N, &alpham1, d_Ax, 1, d_r, 1));
    CUBLASCheck(hipblasDdot(cublasHandle, N, d_r, 1, d_r, 1, &r1));
    
    k = 1;
    while (r1 > tolerance * tolerance && k <= limit) {
        if (k > 1) {
            bb = r1 / r0;
            CUBLASCheck(hipblasDscal(cublasHandle, N, &bb, d_p, 1));
            CUBLASCheck(hipblasDaxpy(cublasHandle, N, &alpha, d_r, 1, d_p, 1));
        } else {
            CUBLASCheck(hipblasDcopy(cublasHandle, N, d_r, 1, d_p, 1));
        }

        CUSPARSECheck(hipsparseSpMV(cusparseHandle, 
            HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecp, &beta, vecAx, 
            HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, buffer));
        CUBLASCheck(hipblasDdot(cublasHandle, N, d_p, 1, d_Ax, 1, &dot));
        a = r1 / dot;

        CUBLASCheck(hipblasDaxpy(cublasHandle, N, &a, d_p, 1, d_x, 1));
        na = -a;
        CUBLASCheck(hipblasDaxpy(cublasHandle, N, &na, d_Ax, 1, d_r, 1));

        r0 = r1;
        CUBLASCheck(hipblasDdot(cublasHandle, N, d_r, 1, d_r, 1, &r1));
        hipDeviceSynchronize();
        
        k++;
    }
    
    CUDACheck(hipMemcpy(h_x, d_x, N * sizeof(double), hipMemcpyDeviceToHost));
    
    std::vector<double> xx(N);
    for (int i = 0; i < N; i++) {
        xx[i] = h_x[i];
    }

    // save iteration info
    iter = k;
    norm = std::sqrt(r1);
    x.setvalues({xx.begin(), xx.end()});

    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);
    
    if (matA) {
        CUSPARSECheck(hipsparseDestroySpMat(matA));
    }
    if (vecx) {
        CUSPARSECheck(hipsparseDestroyDnVec(vecx));
    }
    if (vecAx) {
        CUSPARSECheck(hipsparseDestroyDnVec(vecAx));
    }
    if (vecp) {
        CUSPARSECheck(hipsparseDestroyDnVec(vecp));
    }
    
    free(h_row);
    free(h_col);
    free(h_val);
    free(h_x);
    free(rhs);
    CUDACheck(hipFree(d_col));
    CUDACheck(hipFree(d_row));
    CUDACheck(hipFree(d_val));
    CUDACheck(hipFree(d_x));
    CUDACheck(hipFree(d_r));
    CUDACheck(hipFree(d_p));
    CUDACheck(hipFree(d_Ax));

    return ;

    // auto m_Mat = A.getMat();
	// int num_rows = m_Mat.size(), nnz = 0;
	// int m = num_rows;
	// int num_offsets = m + 1;

    // int*    h_A_rows    = (int*)    malloc(num_offsets * sizeof(int));
    // h_A_rows[0] = 0;
	// for (int i = 0; i < m_Mat.size(); i++) {
	// 	for (int j = 0; j < m_Mat[i].size(); j++) {
	// 		if (m_Mat[i][j].second != 0.f) {
	// 			nnz++;
	// 		}
	// 	}
    //     h_A_rows[i + 1] = nnz;
	// }
	
    // int*    h_A_columns = (int*)    malloc(nnz * sizeof(int));
    // double* h_A_values  = (double*) malloc(nnz * sizeof(double));
    // double* h_X         = (double*) malloc(m * sizeof(double));
	// // from ellpack to csr
    // int cnt = 0;
    // for (int i = 0; i < m_Mat.size(); i++) {
	// 	for (int j = 0; j < m_Mat[i].size(); j++) {
	// 		if (m_Mat[i][j].second != 0.f) {
	// 			h_A_columns[cnt] = m_Mat[i][j].first;
    //             h_A_values[cnt] = m_Mat[i][j].second;
    //             cnt++;
	// 		}
	// 	}
	// }
	// // write A to file
    // // FILE* fpA = fopen("temp/A.txt", "w");
    // // for (int i = 0; i < num_offsets; i++) {
    // //     fprintf(fpA, "%d ", h_A_rows[i]);
    // // }
    // // fprintf(fpA, "\n");
    // // for (int i = 0; i < nnz; i++) {
    // //     fprintf(fpA, "%d %10.10f\n", h_A_columns[i], h_A_values[i]);
    // // }
    // for (int i = 0; i < num_rows; i++)
    //     h_X[i] = 1.0;
    // //--------------------------------------------------------------------------
    // // ### Device memory management ###
    // int*    d_A_rows, *d_A_columns;
    // double* d_A_values;
    // double* h_P = (double*) malloc(m * sizeof(double));
    // Vec     d_B, d_X, d_R, d_P, d_T;

    // // allocate device memory for CSR matrices
    // CUDACheck( hipMalloc((void**) &d_A_rows,    num_offsets * sizeof(int)) );
    // CUDACheck( hipMalloc((void**) &d_A_columns, nnz * sizeof(int)) );
    // CUDACheck( hipMalloc((void**) &d_A_values,  nnz * sizeof(double)) );
    
    // CUDACheck( hipMalloc((void**) &d_B.ptr,     m * sizeof(double)) );
    // CUDACheck( hipMalloc((void**) &d_X.ptr,     m * sizeof(double)) );
    // CUDACheck( hipMalloc((void**) &d_R.ptr,     m * sizeof(double)) );
    // CUDACheck( hipMalloc((void**) &d_P.ptr,     m * sizeof(double)) );
    // CUDACheck( hipMalloc((void**) &d_T.ptr,     m * sizeof(double)) );
    
    // // copy the CSR matrices and vectors into device memory
    // CUDACheck( hipMemcpy(d_A_rows, h_A_rows, num_offsets * sizeof(int),
    //                        hipMemcpyHostToDevice) );
    // CUDACheck( hipMemcpy(d_A_columns, h_A_columns, nnz *  sizeof(int),
    //                        hipMemcpyHostToDevice) );
    // CUDACheck( hipMemcpy(d_A_values, h_A_values, nnz * sizeof(double),
    //                        hipMemcpyHostToDevice) );
    // CUDACheck( hipMemcpy(d_X.ptr, h_X, m * sizeof(double),
    //                        hipMemcpyHostToDevice) );
    // //--------------------------------------------------------------------------
    // // ### cuSPARSE Handle and descriptors initialization ###
    // // create the test matrix on the host
    // hipblasHandle_t   cublasHandle   = NULL;
    // hipsparseHandle_t cusparseHandle = NULL;
    // CUBLASCheck( hipblasCreate(&cublasHandle) );
    // CUSPARSECheck( hipsparseCreate(&cusparseHandle) );
    // // Create dense vectors
    // CUSPARSECheck( hipsparseCreateDnVec(&d_B.vec,     m, d_B.ptr, HIP_R_64F) );
    // CUSPARSECheck( hipsparseCreateDnVec(&d_X.vec,     m, d_X.ptr, HIP_R_64F) );
    // CUSPARSECheck( hipsparseCreateDnVec(&d_R.vec,     m, d_R.ptr, HIP_R_64F) );
    // CUSPARSECheck( hipsparseCreateDnVec(&d_P.vec,   m, d_P.ptr,   HIP_R_64F) );
    // CUSPARSECheck( hipsparseCreateDnVec(&d_T.vec,   m, d_T.ptr,   HIP_R_64F) );
    
    // // copy b
    // auto b_vec = b.generateScalar();
    // CUDACheck( hipMemcpy(d_B.ptr, b_vec.data(), m * sizeof(double),
    //                        hipMemcpyHostToDevice) );

    // hipsparseIndexBase_t  baseIdx = HIPSPARSE_INDEX_BASE_ZERO;
    // hipsparseSpMatDescr_t matA;
    // // A
    // CUSPARSECheck( hipsparseCreateCsr(&matA, m, m, nnz, d_A_rows,
    //                                   d_A_columns, d_A_values,
    //                                   HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
    //                                   baseIdx, HIP_R_64F) );
    
    // // ### Preparation ### 
    // const double Alpha = 0.75;
    // size_t       bufferSizeMV;
    // void*        d_bufferMV;
    // double       Beta = 0.0;
    // CUSPARSECheck( hipsparseSpMV_bufferSize(
    //                     cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
    //                     &Alpha, matA, d_X.vec, &Beta, d_B.vec, HIP_R_64F,
    //                     HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSizeMV) );
    // CUDACheck( hipMalloc(&d_bufferMV, bufferSizeMV) );

    // // X0 = 0
    // CUDACheck( hipMemset(d_X.ptr, 0x0, m * sizeof(double)) );
    // //--------------------------------------------------------------------------
    // // ### Run CG computation ###
    // const double zero      = 0.0;
    // const double one       = 1.0;
    // const double minus_one = -1.0;
    // //--------------------------------------------------------------------------
    // // ### 1 ### R0 = b - A * X0 (using initial guess in X)
    // //    (a) copy b in R0
    // CUDACheck( hipMemcpy(d_R.ptr, d_B.ptr, m * sizeof(double),
    //                        hipMemcpyDeviceToDevice) );
    // //    (b) compute R = -A * X0 + R
    // CUSPARSECheck( hipsparseSpMV(cusparseHandle,
    //                              HIPSPARSE_OPERATION_NON_TRANSPOSE,
    //                              &minus_one, matA, d_X.vec, &one, d_R.vec,
    //                              HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT,
    //                              d_bufferMV) );
    // //--------------------------------------------------------------------------
    // // ### 2 ### P0 = R0
    // CUDACheck( hipMemcpy(d_P.ptr, d_R.ptr, m * sizeof(double),
    //                        hipMemcpyDeviceToDevice) );
    // //--------------------------------------------------------------------------
    // // nrm_R0 = ||R||
    // iter = 0;
    // norm = 1000;
    // double nrm_R;
    // CUBLASCheck( hipblasDnrm2(cublasHandle, m, d_R.ptr, 1, &nrm_R) );
    // double threshold = tolerance * nrm_R;
    // printf("  Initial Residual: Norm %e' threshold %e\n", nrm_R, threshold);
    // //--------------------------------------------------------------------------
    // double delta;
    // CUBLASCheck( hipblasDdot(cublasHandle, m, d_R.ptr, 1, d_R.ptr, 1, &delta) );
    // //--------------------------------------------------------------------------
    // // ### 3 ### repeat until convergence based on max iterations and
    // //           and relative residual
    // // write iterative info into file
    // // FILE* fp = fopen("temp/cg_info.txt", "w");
    // // fprintf(fp, "Initial Residual: Norm %e' threshold %e\n", nrm_R, threshold);
    // // fprintf(fp, "Iteration\tResidual\n");
    // // fprintf(fp, "%d\t%e\n", 0, nrm_R);

    // // FILE* fpP = fopen("temp/cg_P.txt", "w");
    // CUDACheck( hipMemcpy(h_P, d_P.ptr, m * sizeof(double),
    //                        hipMemcpyDeviceToHost) );
    // // for (int i = 0; i < m; i++) {
    // //     fprintf(fpP, "%e\t", h_P[i]);
    // //     if (i == m - 1) {
    // //         fprintf(fpP, "\n");
    // //     }
    // // }
    // // for (int i = 0; i < limit; i++) {
    // while (iter < limit && nrm_R > threshold) {
    //     // printf("  Iteration = %d; Error Norm = %e\n", i, nrm_R);
    //     //----------------------------------------------------------------------
    //     // ### 4 ### alpha = (R_i, R_i) / (A * P_i, P_i)
    //     //     (a) T  = A * P_i
    //     CUSPARSECheck( hipsparseSpMV(cusparseHandle,
    //                                  HIPSPARSE_OPERATION_NON_TRANSPOSE, &one,
    //                                  matA, d_P.vec, &zero, d_T.vec,
    //                                  HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT,
    //                                  d_bufferMV) );
    //     //     (b) denominator = (T, P_i)
    //     double denominator;
    //     CUBLASCheck( hipblasDdot(cublasHandle, m, d_T.ptr, 1, d_P.ptr, 1,
    //                              &denominator) );
    //     //     (c) alpha = delta / denominator
    //     double alpha = delta / denominator;
    //     // PRINT_INFO(delta)
    //     // PRINT_INFO(denominator)
    //     // PRINT_INFO(alpha)
    //     //----------------------------------------------------------------------
    //     // ### 6 ###  X_i+1 = X_i + alpha * P
    //     //    (a) X_i+1 = -alpha * T + X_i
    //     CUBLASCheck( hipblasDaxpy(cublasHandle, m, &alpha, d_P.ptr, 1,
    //                               d_X.ptr, 1) );
    //     //----------------------------------------------------------------------
    //     // ### 7 ###  R_i+1 = R_i - alpha * (A * P)
    //     //    (a) R_i+1 = -alpha * T + R_i
    //     double minus_alpha = -alpha;
    //     CUBLASCheck( hipblasDaxpy(cublasHandle, m, &minus_alpha, d_T.ptr, 1,
    //                               d_R.ptr, 1) );
    //     //----------------------------------------------------------------------
    //     // ### 8 ###  check ||R_i+1|| < threshold
    //     CUBLASCheck( hipblasDnrm2(cublasHandle, m, d_R.ptr, 1, &nrm_R) );
    //     // fprintf(fp, "%d\t%e\n", iter + 1, nrm_R);
    //     iter++;
    //     if (nrm_R < threshold)
    //         break;
    //     //----------------------------------------------------------------------
    //     // ### 8 ### beta = (R_i+1, R_i+1) / (R_i, R_i)
    //     //    (a) delta_new => (R_i+1, R_i+1)
    //     double delta_new;
    //     CUBLASCheck( hipblasDdot(cublasHandle, m, d_R.ptr, 1, d_R.ptr, 1,
    //                              &delta_new) );
    //     //    (b) beta => delta_new / delta
    //     double beta = delta_new / delta;
    //     delta       = delta_new;
    //     //----------------------------------------------------------------------
    //     // ### 9 ###  P_i+1 = R_i+1 + beta * P_i
    //     //    (a) copy R_i+1 in P_i
    //     CUDACheck( hipMemcpy(d_P.ptr, d_R.ptr, m * sizeof(double),
    //                            hipMemcpyDeviceToDevice) );
    //     //    (b) P_i+1 = beta * P_i + R_i+1
    //     CUBLASCheck( hipblasDaxpy(cublasHandle, m, &beta, d_P.ptr, 1,
    //                               d_P.ptr, 1) );
    //     CUDACheck( hipMemcpy(h_P, d_P.ptr, m * sizeof(double),
    //                            hipMemcpyDeviceToHost) );
    //     // for (int i = 0; i < m; i++) {
    //     //     fprintf(fpP, "%e\t", h_P[i]);
    //     //     if (i == m - 1) {
    //     //         fprintf(fpP, "\n");
    //     //     }
    //     // }
    // }
    // //--------------------------------------------------------------------------
    // // printf("Check Solution\n"); // ||R = b - A * X||
    // //    (a) copy b in R
    // CUDACheck( hipMemcpy(d_R.ptr, d_B.ptr, m * sizeof(double),
    //                        hipMemcpyDeviceToDevice) );
    // // R = -A * X + R
    // CUSPARSECheck( hipsparseSpMV(cusparseHandle,
    //                              HIPSPARSE_OPERATION_NON_TRANSPOSE, &minus_one,
    //                              matA, d_X.vec, &one, d_R.vec, HIP_R_64F,
    //                              HIPSPARSE_SPMV_ALG_DEFAULT, d_bufferMV) );
    // // check ||R||
    // CUBLASCheck( hipblasDnrm2(cublasHandle, m, d_R.ptr, 1, &nrm_R) );
    // // copy result
    // CUDACheck( hipMemcpy(h_X, d_X.ptr, m * sizeof(double),
    //                        hipMemcpyDeviceToHost) );
    // std::vector<Scalar> xx(m);
    // for (int i = 0; i < m; i++) {
    //     xx[i] = h_X[i];
    // }
    
    // norm = nrm_R;// * tolerance;
    // printf("Final iterations: %d error norm = %e\n", iter, norm);
    
    // //--------------------------------------------------------------------------
    // x.setvalues({xx.begin(), xx.end()});

    // //--------------------------------------------------------------------------
    // // ### Free resources ###
    // CUSPARSECheck( hipsparseDestroyDnVec(d_B.vec) );
    // CUSPARSECheck( hipsparseDestroyDnVec(d_X.vec) );
    // CUSPARSECheck( hipsparseDestroyDnVec(d_R.vec) );
    // CUSPARSECheck( hipsparseDestroyDnVec(d_P.vec) );
    // CUSPARSECheck( hipsparseDestroyDnVec(d_T.vec) );
    // CUSPARSECheck( hipsparseDestroySpMat(matA) );
    // CUSPARSECheck( hipsparseDestroy(cusparseHandle) );
    // CUBLASCheck( hipblasDestroy(cublasHandle) );

    // free(h_A_rows);
    // free(h_A_columns);
    // free(h_A_values);
    // free(h_X);

    // CUDACheck( hipFree(d_X.ptr) );
    // CUDACheck( hipFree(d_B.ptr) );
    // CUDACheck( hipFree(d_R.ptr) );
    // CUDACheck( hipFree(d_P.ptr) );
    // CUDACheck( hipFree(d_T.ptr) );
    // CUDACheck( hipFree(d_A_values) );
    // CUDACheck( hipFree(d_A_columns) );
    // CUDACheck( hipFree(d_A_rows) );
    // CUDACheck( hipFree(d_bufferMV) );
    // return ;
}

void PCG_ICC(const SymetrixSparseMatrix& A,Vector& x,const Vector& b,double tolerance,int limit,int& iter,double& norm)
{
    printf("PCG_ICC...\n");
    auto m_Mat = A.getMat();
	int num_rows = m_Mat.size(), nnz = 0;
	int m = num_rows;
	int num_offsets = m + 1;

    int*    h_A_rows    = (int*)    malloc(num_offsets * sizeof(int));
    h_A_rows[0] = 0;
	for (int i = 0; i < m_Mat.size(); i++) {
		for (int j = 0; j < m_Mat[i].size(); j++) {
			if (m_Mat[i][j].second != 0) {
				nnz++;
			}
		}
        h_A_rows[i + 1] = nnz;
	}
	
    int*    h_A_columns = (int*)    malloc(nnz * sizeof(int));
    double* h_A_values  = (double*) malloc(nnz * sizeof(double));
    double* h_X         = (double*) malloc(m * sizeof(double));
	// from ellpack to csr
    int cnt = 0;
    for (int i = 0; i < m_Mat.size(); i++) {
		for (int j = 0; j < m_Mat[i].size(); j++) {
			if (m_Mat[i][j].second != 0) {
				h_A_columns[cnt] = m_Mat[i][j].first;
                h_A_values[cnt] = m_Mat[i][j].second;
                cnt++;
			}
		}
	}
	
    for (int i = 0; i < num_rows; i++)
        h_X[i] = 1.0;
    //--------------------------------------------------------------------------
    // ### Device memory management ###
    int*    d_A_rows, *d_A_columns;
    double* d_A_values, *d_L_values;
    // R_aux = z
    Vec     d_B, d_X, d_R, d_R_aux, d_P, d_T, d_tmp;

    // allocate device memory for CSR matrices
    CUDACheck( hipMalloc((void**) &d_A_rows,    num_offsets * sizeof(int)) );
    CUDACheck( hipMalloc((void**) &d_A_columns, nnz * sizeof(int)) );
    CUDACheck( hipMalloc((void**) &d_A_values,  nnz * sizeof(double)) );
    CUDACheck( hipMalloc((void**) &d_L_values,  nnz * sizeof(double)) );

    CUDACheck( hipMalloc((void**) &d_B.ptr,     m * sizeof(double)) );
    CUDACheck( hipMalloc((void**) &d_X.ptr,     m * sizeof(double)) );
    CUDACheck( hipMalloc((void**) &d_R.ptr,     m * sizeof(double)) );
    CUDACheck( hipMalloc((void**) &d_R_aux.ptr, m * sizeof(double)) );
    CUDACheck( hipMalloc((void**) &d_P.ptr,     m * sizeof(double)) );
    CUDACheck( hipMalloc((void**) &d_T.ptr,     m * sizeof(double)) );
    CUDACheck( hipMalloc((void**) &d_tmp.ptr,   m * sizeof(double)) );

    // copy the CSR matrices and vectors into device memory
    CUDACheck( hipMemcpy(d_A_rows, h_A_rows, num_offsets * sizeof(int),
                           hipMemcpyHostToDevice) );
    CUDACheck( hipMemcpy(d_A_columns, h_A_columns, nnz *  sizeof(int),
                           hipMemcpyHostToDevice) );
    CUDACheck( hipMemcpy(d_A_values, h_A_values, nnz * sizeof(double),
                           hipMemcpyHostToDevice) );
    CUDACheck( hipMemcpy(d_L_values, h_A_values, nnz * sizeof(double),
                           hipMemcpyHostToDevice) );
    CUDACheck( hipMemcpy(d_X.ptr, h_X, m * sizeof(double),
                           hipMemcpyHostToDevice) );
    //--------------------------------------------------------------------------
    // ### cuSPARSE Handle and descriptors initialization ###
    // create the test matrix on the host
    hipblasHandle_t   cublasHandle   = NULL;
    hipsparseHandle_t cusparseHandle = NULL;
    CUBLASCheck( hipblasCreate(&cublasHandle) );
    CUSPARSECheck( hipsparseCreate(&cusparseHandle) );
    // Create dense vectors
    CUSPARSECheck( hipsparseCreateDnVec(&d_B.vec,     m, d_B.ptr, HIP_R_64F) );
    CUSPARSECheck( hipsparseCreateDnVec(&d_X.vec,     m, d_X.ptr, HIP_R_64F) );
    CUSPARSECheck( hipsparseCreateDnVec(&d_R.vec,     m, d_R.ptr, HIP_R_64F) );
    CUSPARSECheck( hipsparseCreateDnVec(&d_R_aux.vec, m, d_R_aux.ptr,
                                        HIP_R_64F) );
    CUSPARSECheck( hipsparseCreateDnVec(&d_P.vec,   m, d_P.ptr,   HIP_R_64F) );
    CUSPARSECheck( hipsparseCreateDnVec(&d_T.vec,   m, d_T.ptr,   HIP_R_64F) );
    CUSPARSECheck( hipsparseCreateDnVec(&d_tmp.vec, m, d_tmp.ptr, HIP_R_64F) );

    // copy b
    auto b_vec = b.generateScalar();
    CUDACheck( hipMemcpy(d_B.ptr, b_vec.data(), m * sizeof(double),
                           hipMemcpyHostToDevice) );

    hipsparseIndexBase_t  baseIdx = HIPSPARSE_INDEX_BASE_ZERO;
    hipsparseSpMatDescr_t matA, matL;
    int*                 d_L_rows      = d_A_rows;
    int*                 d_L_columns   = d_A_columns;
    hipsparseFillMode_t   fill_lower    = HIPSPARSE_FILL_MODE_LOWER;
    hipsparseDiagType_t   diag_non_unit = HIPSPARSE_DIAG_TYPE_NON_UNIT;
    // A
    CUSPARSECheck( hipsparseCreateCsr(&matA, m, m, nnz, d_A_rows,
                                      d_A_columns, d_A_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      baseIdx, HIP_R_64F) );
    // L
    CUSPARSECheck( hipsparseCreateCsr(&matL, m, m, nnz, d_L_rows,
                                      d_L_columns, d_L_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      baseIdx, HIP_R_64F) );
    CUSPARSECheck( hipsparseSpMatSetAttribute(matL,
                                              HIPSPARSE_SPMAT_FILL_MODE,
                                              &fill_lower, sizeof(fill_lower)) );
    CUSPARSECheck( hipsparseSpMatSetAttribute(matL,
                                              HIPSPARSE_SPMAT_DIAG_TYPE,
                                              &diag_non_unit,
                                              sizeof(diag_non_unit)) );
    //--------------------------------------------------------------------------
    // ### Preparation ### b = A * X
    // from here
    const double Alpha = 0.75;
    size_t       bufferSizeMV;
    void*        d_bufferMV;
    double       Beta = 0.0;
    CUSPARSECheck( hipsparseSpMV_bufferSize(
                        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &Alpha, matA, d_X.vec, &Beta, d_B.vec, HIP_R_64F,
                        HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSizeMV) );
    CUDACheck( hipMalloc(&d_bufferMV, bufferSizeMV) );

    // CUSPARSECheck( hipsparseSpMV(
    //                     cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
    //                     &Alpha, matA, d_X.vec, &Beta, d_B.vec, HIP_R_64F,
    //                     HIPSPARSE_SPMV_ALG_DEFAULT, d_bufferMV) );
    
    // to here, maybe needed deletion or modifaction

    // X0 = 0
    CUDACheck( hipMemset(d_X.ptr, 0x0, m * sizeof(double)) );
    //--------------------------------------------------------------------------
    // Perform Incomplete-Cholesky factorization of A (csric0) -> L, L^T
    hipsparseMatDescr_t descrM;
    csric02Info_t      infoM        = NULL;
    int                bufferSizeIC = 0;
    void*              d_bufferIC;
    CUSPARSECheck( hipsparseCreateMatDescr(&descrM) );
    CUSPARSECheck( hipsparseSetMatIndexBase(descrM, baseIdx) );
    CUSPARSECheck( hipsparseSetMatType(descrM, HIPSPARSE_MATRIX_TYPE_GENERAL) );
    CUSPARSECheck( hipsparseSetMatFillMode(descrM, HIPSPARSE_FILL_MODE_LOWER) );
    CUSPARSECheck( hipsparseSetMatDiagType(descrM,
                                           HIPSPARSE_DIAG_TYPE_NON_UNIT) );
    CUSPARSECheck( hipsparseCreateCsric02Info(&infoM) );

    CUSPARSECheck( hipsparseDcsric02_bufferSize(
                        cusparseHandle, m, nnz, descrM, d_L_values,
                        d_A_rows, d_A_columns, infoM, &bufferSizeIC) );
    CUDACheck( hipMalloc(&d_bufferIC, bufferSizeIC) );
    CUSPARSECheck( hipsparseDcsric02_analysis(
                        cusparseHandle, m, nnz, descrM, d_L_values,
                        d_A_rows, d_A_columns, infoM,
                        HIPSPARSE_SOLVE_POLICY_NO_LEVEL, d_bufferIC) );
    int structural_zero;
    CUSPARSECheck( hipsparseXcsric02_zeroPivot(cusparseHandle, infoM,
                                               &structural_zero) );
    // M = L * L^T
    CUSPARSECheck( hipsparseDcsric02(
                        cusparseHandle, m, nnz, descrM, d_L_values,
                        d_A_rows, d_A_columns, infoM,
                        HIPSPARSE_SOLVE_POLICY_NO_LEVEL, d_bufferIC) );
    // Find numerical zero
    int numerical_zero;
    CUSPARSECheck( hipsparseXcsric02_zeroPivot(cusparseHandle, infoM,
                                               &numerical_zero) );

    CUSPARSECheck( hipsparseDestroyCsric02Info(infoM) );
    CUSPARSECheck( hipsparseDestroyMatDescr(descrM) );
    CUDACheck( hipFree(d_bufferIC) );
    //--------------------------------------------------------------------------
    // ### Run CG computation ###
    // printf("CG loop:\n");
    // gpu_CG(cublasHandle, cusparseHandle, m,
    //        matA, matL, d_B, d_X, d_R, d_R_aux, d_P, d_T,
    //        d_tmp, d_bufferMV, maxIterations, tolerance);

    const double zero      = 0.0;
    const double one       = 1.0;
    const double minus_one = -1.0;
    //--------------------------------------------------------------------------
    // ### 1 ### R0 = b - A * X0 (using initial guess in X)
    //    (a) copy b in R0
    CUDACheck( hipMemcpy(d_R.ptr, d_B.ptr, m * sizeof(double),
                           hipMemcpyDeviceToDevice) );
    //    (b) compute R = -A * X0 + R
    CUSPARSECheck( hipsparseSpMV(cusparseHandle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &minus_one, matA, d_X.vec, &one, d_R.vec,
                                 HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT,
                                 d_bufferMV) );
    //--------------------------------------------------------------------------
    // ### 2 ### R_i_aux = L^-1 L^-T R_i
    size_t              bufferSizeL, bufferSizeLT;
    void*               d_bufferL, *d_bufferLT;
    hipsparseSpSVDescr_t spsvDescrL, spsvDescrLT;
    //    (a) L^-T tmp => R_i_aux    (triangular solver)
    CUSPARSECheck( hipsparseSpSV_createDescr(&spsvDescrLT) );
    CUSPARSECheck( hipsparseSpSV_bufferSize(
                        cusparseHandle, HIPSPARSE_OPERATION_TRANSPOSE,
                        &one, matL, d_R.vec, d_tmp.vec, HIP_R_64F,
                        HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrLT, &bufferSizeLT) );
    CUDACheck( hipMalloc(&d_bufferLT, bufferSizeLT) );
    CUSPARSECheck( hipsparseSpSV_analysis(
                        cusparseHandle, HIPSPARSE_OPERATION_TRANSPOSE,
                        &one, matL, d_R.vec, d_tmp.vec, HIP_R_64F,
                        HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrLT, d_bufferLT) );
    CUDACheck( hipMemset(d_tmp.ptr, 0x0, m * sizeof(double)) );
    CUSPARSECheck( hipsparseSpSV_solve(
                        cusparseHandle, HIPSPARSE_OPERATION_TRANSPOSE,
                        &one, matL, d_R.vec, d_tmp.vec, HIP_R_64F,
                        HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrLT) );

    //    (b) L^-T R_i => tmp    (triangular solver)
    CUSPARSECheck( hipsparseSpSV_createDescr(&spsvDescrL) );
    CUSPARSECheck( hipsparseSpSV_bufferSize(
                        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &one, matL, d_tmp.vec, d_R_aux.vec, HIP_R_64F,
                        HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, &bufferSizeL) );
    CUDACheck( hipMalloc(&d_bufferL, bufferSizeL) );
    CUSPARSECheck( hipsparseSpSV_analysis(
                        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &one, matL, d_tmp.vec, d_R_aux.vec, HIP_R_64F,
                        HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, d_bufferL) );
    CUDACheck( hipMemset(d_R_aux.ptr, 0x0, m * sizeof(double)) );
    CUSPARSECheck( hipsparseSpSV_solve(
                        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &one, matL, d_tmp.vec, d_R_aux.vec, HIP_R_64F,
                        HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL) );
    //--------------------------------------------------------------------------
    // ### 3 ### P0 = R0_aux
    CUDACheck( hipMemcpy(d_P.ptr, d_R_aux.ptr, m * sizeof(double),
                           hipMemcpyDeviceToDevice) );
    //--------------------------------------------------------------------------
    // nrm_R0 = ||R||
    double nrm_R;
    CUBLASCheck( hipblasDnrm2(cublasHandle, m, d_R.ptr, 1, &nrm_R) );
    double threshold = tolerance * nrm_R;
    printf("  Initial Residual: Norm %e' threshold %e\n", nrm_R, threshold);
    //--------------------------------------------------------------------------
    double delta;
    CUBLASCheck( hipblasDdot(cublasHandle, m, d_R.ptr, 1, d_R.ptr, 1, &delta) );
    //--------------------------------------------------------------------------
    // ### 4 ### repeat until convergence based on max iterations and
    //           and relative residual
    for (int i = 0; i < limit; i++) {
        // printf("  Iteration = %d; Error Norm = %e\n", i, nrm_R);
        //----------------------------------------------------------------------
        // ### 5 ### alpha = (R_i, R_aux_i) / (A * P_i, P_i)
        //     (a) T  = A * P_i
        CUSPARSECheck( hipsparseSpMV(cusparseHandle,
                                     HIPSPARSE_OPERATION_NON_TRANSPOSE, &one,
                                     matA, d_P.vec, &zero, d_T.vec,
                                     HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT,
                                     d_bufferMV) );
        //     (b) denominator = (T, P_i)
        double denominator;
        CUBLASCheck( hipblasDdot(cublasHandle, m, d_T.ptr, 1, d_P.ptr, 1,
                                 &denominator) );
        //     (c) alpha = delta / denominator
        double alpha = delta / denominator;
        // PRINT_INFO(delta)
        // PRINT_INFO(denominator)
        // PRINT_INFO(alpha)
        //----------------------------------------------------------------------
        // ### 6 ###  X_i+1 = X_i + alpha * P
        //    (a) X_i+1 = -alpha * T + X_i
        CUBLASCheck( hipblasDaxpy(cublasHandle, m, &alpha, d_P.ptr, 1,
                                  d_X.ptr, 1) );
        //----------------------------------------------------------------------
        // ### 7 ###  R_i+1 = R_i - alpha * (A * P)
        //    (a) R_i+1 = -alpha * T + R_i
        double minus_alpha = -alpha;
        CUBLASCheck( hipblasDaxpy(cublasHandle, m, &minus_alpha, d_T.ptr, 1,
                                  d_R.ptr, 1) );
        //----------------------------------------------------------------------
        // ### 8 ###  check ||R_i+1|| < threshold
        CUBLASCheck( hipblasDnrm2(cublasHandle, m, d_R.ptr, 1, &nrm_R) );
        // PRINT_INFO(nrm_R)
        iter++;
        if (nrm_R < threshold)
            break;
        //----------------------------------------------------------------------
        // ### 9 ### R_aux_i+1 = L^-1 L^-T R_i+1
        //    (a) L^-T R_i+1 => tmp    (triangular solver)
        CUDACheck( hipMemset(d_tmp.ptr,   0x0, m * sizeof(double)) );
        CUDACheck( hipMemset(d_R_aux.ptr, 0x0, m * sizeof(double)) );
        CUSPARSECheck( hipsparseSpSV_solve(cusparseHandle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &one, matL, d_R.vec, d_tmp.vec,
                                           HIP_R_64F,
                                           HIPSPARSE_SPSV_ALG_DEFAULT,
                                           spsvDescrL) );
        //    (b) L^-T tmp => R_aux_i+1    (triangular solver)
        CUSPARSECheck( hipsparseSpSV_solve(cusparseHandle,
                                           HIPSPARSE_OPERATION_TRANSPOSE,
                                           &one, matL, d_tmp.vec,
                                           d_R_aux.vec, HIP_R_64F,
                                           HIPSPARSE_SPSV_ALG_DEFAULT,
                                           spsvDescrLT) );
        //----------------------------------------------------------------------
        // ### 10 ### beta = (R_i+1, R_aux_i+1) / (R_i, R_aux_i)
        //    (a) delta_new => (R_i+1, R_aux_i+1)
        double delta_new;
        CUBLASCheck( hipblasDdot(cublasHandle, m, d_R.ptr, 1, d_R_aux.ptr, 1,
                                 &delta_new) );
        //    (b) beta => delta_new / delta
        double beta = delta_new / delta;
        // PRINT_INFO(delta_new)
        // PRINT_INFO(beta)
        delta       = delta_new;
        //----------------------------------------------------------------------
        // ### 11 ###  P_i+1 = R_aux_i+1 + beta * P_i
        //    (a) copy R_aux_i+1 in P_i
        CUDACheck( hipMemcpy(d_P.ptr, d_R_aux.ptr, m * sizeof(double),
                               hipMemcpyDeviceToDevice) );
        //    (b) P_i+1 = beta * P_i + R_aux_i+1
        CUBLASCheck( hipblasDaxpy(cublasHandle, m, &beta, d_P.ptr, 1,
                                  d_P.ptr, 1) );
    }
    //--------------------------------------------------------------------------
    // printf("Check Solution\n"); // ||R = b - A * X||
    //    (a) copy b in R
    CUDACheck( hipMemcpy(d_R.ptr, d_B.ptr, m * sizeof(double),
                           hipMemcpyDeviceToDevice) );
    // R = -A * X + R
    CUSPARSECheck( hipsparseSpMV(cusparseHandle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE, &minus_one,
                                 matA, d_X.vec, &one, d_R.vec, HIP_R_64F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, d_bufferMV) );
    // check ||R||
    CUBLASCheck( hipblasDnrm2(cublasHandle, m, d_R.ptr, 1, &nrm_R) );
    // copy result
    CUDACheck( hipMemcpy(h_X, d_X.ptr, m * sizeof(double),
                           hipMemcpyDeviceToHost) );
    std::vector<Scalar> xx(m);
    for (int i = 0; i < m; i++) {
        xx[i] = h_X[i];
    }
    
    norm = nrm_R * tolerance;
    // printf("Final iterations: %d error norm = %e\n", iter, norm);
    
    //--------------------------------------------------------------------------
    CUSPARSECheck( hipsparseSpSV_destroyDescr(spsvDescrL) );
    CUSPARSECheck( hipsparseSpSV_destroyDescr(spsvDescrLT) );
    CUDACheck( hipFree(d_bufferL) );
    CUDACheck( hipFree(d_bufferLT) );

    x.setvalues({xx.begin(), xx.end()});

    //--------------------------------------------------------------------------
    // ### Free resources ###
    CUSPARSECheck( hipsparseDestroyDnVec(d_B.vec) );
    CUSPARSECheck( hipsparseDestroyDnVec(d_X.vec) );
    CUSPARSECheck( hipsparseDestroyDnVec(d_R.vec) );
    CUSPARSECheck( hipsparseDestroyDnVec(d_R_aux.vec) );
    CUSPARSECheck( hipsparseDestroyDnVec(d_P.vec) );
    CUSPARSECheck( hipsparseDestroyDnVec(d_T.vec) );
    CUSPARSECheck( hipsparseDestroyDnVec(d_tmp.vec) );
    CUSPARSECheck( hipsparseDestroySpMat(matA) );
    CUSPARSECheck( hipsparseDestroySpMat(matL) );
    CUSPARSECheck( hipsparseDestroy(cusparseHandle) );
    CUBLASCheck( hipblasDestroy(cublasHandle) );

    free(h_A_rows);
    free(h_A_columns);
    free(h_A_values);
    free(h_X);

    CUDACheck( hipFree(d_X.ptr) );
    CUDACheck( hipFree(d_B.ptr) );
    CUDACheck( hipFree(d_R.ptr) );
    CUDACheck( hipFree(d_R_aux.ptr) );
    CUDACheck( hipFree(d_P.ptr) );
    CUDACheck( hipFree(d_T.ptr) );
    CUDACheck( hipFree(d_tmp.ptr) );
    CUDACheck( hipFree(d_A_values) );
    CUDACheck( hipFree(d_A_columns) );
    CUDACheck( hipFree(d_A_rows) );
    CUDACheck( hipFree(d_L_values) );
    CUDACheck( hipFree(d_bufferMV) );
    return ;
}

void BiCGSTAB(const SymetrixSparseMatrix& A,Vector& x,const Vector& b,double tolerance,int limit,int& iter,double& norm)
{
    printf("BiCGSTAB...\n");
    auto m_Mat = A.getMat();
	int num_rows = m_Mat.size(), nnz = 0;
	int m = num_rows;
	int num_offsets = m + 1;

    int*    h_A_rows    = (int*)    malloc(num_offsets * sizeof(int));
    h_A_rows[0] = 0;
	for (int i = 0; i < m_Mat.size(); i++) {
		for (int j = 0; j < m_Mat[i].size(); j++) {
			if (m_Mat[i][j].second != 0) {
				nnz++;
			}
		}
        h_A_rows[i + 1] = nnz;
	}
	
    int*    h_A_columns = (int*)    malloc(nnz * sizeof(int));
    double* h_A_values  = (double*) malloc(nnz * sizeof(double));
    double* h_X         = (double*) malloc(m * sizeof(double));
	// from ellpack to csr
    int cnt = 0;
    for (int i = 0; i < m_Mat.size(); i++) {
		for (int j = 0; j < m_Mat[i].size(); j++) {
			if (m_Mat[i][j].second != 0) {
				h_A_columns[cnt] = m_Mat[i][j].first;
                h_A_values[cnt] = m_Mat[i][j].second;
                cnt++;
            }
		}
	}

    // printf("Testing BiCGStab\n");
    for (int i = 0; i < num_rows; i++)
        h_X[i] = 0;
    //--------------------------------------------------------------------------
    // ### Device memory management ###
    int*    d_A_rows, *d_A_columns;
    double* d_A_values, *d_M_values;
    Vec     d_B, d_X, d_R, d_R0, d_P, d_P_aux, d_S, d_S_aux, d_V, d_T, d_tmp;

    // allocate device memory for CSR matrices
    CUDACheck( hipMalloc((void**) &d_A_rows,    num_offsets * sizeof(int)) );
    CUDACheck( hipMalloc((void**) &d_A_columns, nnz * sizeof(int)) );
    CUDACheck( hipMalloc((void**) &d_A_values,  nnz * sizeof(double)) );
    CUDACheck( hipMalloc((void**) &d_M_values,  nnz * sizeof(double)) );

    CUDACheck( hipMalloc((void**) &d_B.ptr,     m * sizeof(double)) );
    CUDACheck( hipMalloc((void**) &d_X.ptr,     m * sizeof(double)) );
    CUDACheck( hipMalloc((void**) &d_R.ptr,     m * sizeof(double)) );
    CUDACheck( hipMalloc((void**) &d_R0.ptr,    m * sizeof(double)) );
    CUDACheck( hipMalloc((void**) &d_P.ptr,     m * sizeof(double)) );
    CUDACheck( hipMalloc((void**) &d_P_aux.ptr, m * sizeof(double)) );
    CUDACheck( hipMalloc((void**) &d_S.ptr,     m * sizeof(double)) );
    CUDACheck( hipMalloc((void**) &d_S_aux.ptr, m * sizeof(double)) );
    CUDACheck( hipMalloc((void**) &d_V.ptr,     m * sizeof(double)) );
    CUDACheck( hipMalloc((void**) &d_T.ptr,     m * sizeof(double)) );
    CUDACheck( hipMalloc((void**) &d_tmp.ptr,   m * sizeof(double)) );

    // copy the CSR matrices and vectors into device memory
    CUDACheck( hipMemcpy(d_A_rows, h_A_rows, num_offsets * sizeof(int),
                           hipMemcpyHostToDevice) );
    CUDACheck( hipMemcpy(d_A_columns, h_A_columns, nnz *  sizeof(int),
                           hipMemcpyHostToDevice) );
    CUDACheck( hipMemcpy(d_A_values, h_A_values, nnz * sizeof(double),
                           hipMemcpyHostToDevice) );
    CUDACheck( hipMemcpy(d_M_values, h_A_values, nnz * sizeof(double),
                           hipMemcpyHostToDevice) );
    CUDACheck( hipMemcpy(d_X.ptr, h_X, m * sizeof(double),
                           hipMemcpyHostToDevice) );
    //--------------------------------------------------------------------------
    // ### cuSPARSE Handle and descriptors initialization ###
    // create the test matrix on the host
    hipblasHandle_t   cublasHandle   = NULL;
    hipsparseHandle_t cusparseHandle = NULL;
    CUBLASCheck( hipblasCreate(&cublasHandle) );
    CUSPARSECheck( hipsparseCreate(&cusparseHandle) );
    // Create dense vectors
    CUSPARSECheck( hipsparseCreateDnVec(&d_B.vec,     m, d_B.ptr, HIP_R_64F) );
    CUSPARSECheck( hipsparseCreateDnVec(&d_X.vec,     m, d_X.ptr, HIP_R_64F) );
    CUSPARSECheck( hipsparseCreateDnVec(&d_R.vec,     m, d_R.ptr, HIP_R_64F) );
    CUSPARSECheck( hipsparseCreateDnVec(&d_R0.vec,    m, d_R0.ptr, HIP_R_64F) );
    CUSPARSECheck( hipsparseCreateDnVec(&d_P.vec,     m, d_P.ptr, HIP_R_64F) );
    CUSPARSECheck( hipsparseCreateDnVec(&d_P_aux.vec, m, d_P_aux.ptr,
                                        HIP_R_64F) );
    CUSPARSECheck( hipsparseCreateDnVec(&d_S.vec,     m, d_S.ptr, HIP_R_64F) );
    CUSPARSECheck( hipsparseCreateDnVec(&d_S_aux.vec, m, d_S_aux.ptr,
                                        HIP_R_64F) );
    CUSPARSECheck( hipsparseCreateDnVec(&d_V.vec,   m, d_V.ptr,   HIP_R_64F) );
    CUSPARSECheck( hipsparseCreateDnVec(&d_T.vec,   m, d_T.ptr,   HIP_R_64F) );
    CUSPARSECheck( hipsparseCreateDnVec(&d_tmp.vec, m, d_tmp.ptr, HIP_R_64F) );

    // copy b
    auto b_vec = b.generateScalar();
    CUDACheck( hipMemcpy(d_B.ptr, b_vec.data(), m * sizeof(double),
                           hipMemcpyHostToDevice) );

    hipsparseIndexBase_t  baseIdx = HIPSPARSE_INDEX_BASE_ZERO;
    // IMPORTANT: Upper/Lower triangular decompositions of A
    //            (matM_lower, matM_upper) must use two distinct descriptors
    hipsparseSpMatDescr_t matA, matM_lower, matM_upper;
    hipsparseMatDescr_t   matLU;
    int*                 d_M_rows      = d_A_rows;
    int*                 d_M_columns   = d_A_columns;
    hipsparseFillMode_t   fill_lower    = HIPSPARSE_FILL_MODE_LOWER;
    hipsparseDiagType_t   diag_unit     = HIPSPARSE_DIAG_TYPE_UNIT;
    hipsparseFillMode_t   fill_upper    = HIPSPARSE_FILL_MODE_UPPER;
    hipsparseDiagType_t   diag_non_unit = HIPSPARSE_DIAG_TYPE_NON_UNIT;
    // A
    CUSPARSECheck( hipsparseCreateCsr(&matA, m, m, nnz, d_A_rows,
                                      d_A_columns, d_A_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      baseIdx, HIP_R_64F) );
    // M_lower
    CUSPARSECheck( hipsparseCreateCsr(&matM_lower, m, m, nnz, d_M_rows,
                                      d_M_columns, d_M_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      baseIdx, HIP_R_64F) );
    CUSPARSECheck( hipsparseSpMatSetAttribute(matM_lower,
                                              HIPSPARSE_SPMAT_FILL_MODE,
                                              &fill_lower, sizeof(fill_lower)) );
    CUSPARSECheck( hipsparseSpMatSetAttribute(matM_lower,
                                              HIPSPARSE_SPMAT_DIAG_TYPE,
                                              &diag_unit, sizeof(diag_unit)) );
    // M_upper
    CUSPARSECheck( hipsparseCreateCsr(&matM_upper, m, m, nnz, d_M_rows,
                                      d_M_columns, d_M_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      baseIdx, HIP_R_64F) );
    CUSPARSECheck( hipsparseSpMatSetAttribute(matM_upper,
                                              HIPSPARSE_SPMAT_FILL_MODE,
                                              &fill_upper, sizeof(fill_upper)) );
    CUSPARSECheck( hipsparseSpMatSetAttribute(matM_upper,
                                              HIPSPARSE_SPMAT_DIAG_TYPE,
                                              &diag_non_unit,
                                              sizeof(diag_non_unit)) );
    //--------------------------------------------------------------------------
    // ### Preparation ### b = A * X
    const double Alpha = 0.75;
    size_t bufferSizeMV;
    void*  d_bufferMV;
    double beta = 0.0;
    CUSPARSECheck( hipsparseSpMV_bufferSize(
                        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &Alpha, matA, d_X.vec, &beta, d_B.vec, HIP_R_64F,
                        HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSizeMV) );
    CUDACheck( hipMalloc(&d_bufferMV, bufferSizeMV) );

    // CUSPARSECheck( hipsparseSpMV(
    //                     cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
    //                     &Alpha, matA, d_X.vec, &beta, d_B.vec, HIP_R_64F,
    //                     HIPSPARSE_SPMV_ALG_DEFAULT, d_bufferMV) );
    // X0 = 0
    CUDACheck( hipMemset(d_X.ptr, 0x0, m * sizeof(double)) );
    //--------------------------------------------------------------------------
    // Perform Incomplete-LU factorization of A (csrilu0) -> M_lower, M_upper
    csrilu02Info_t infoM        = NULL;
    int            bufferSizeLU = 0;
    void*          d_bufferLU;
    CUSPARSECheck( hipsparseCreateMatDescr(&matLU) );
    CUSPARSECheck( hipsparseSetMatType(matLU, HIPSPARSE_MATRIX_TYPE_GENERAL) );
    CUSPARSECheck( hipsparseSetMatIndexBase(matLU, baseIdx) );
    CUSPARSECheck( hipsparseCreateCsrilu02Info(&infoM) );

    CUSPARSECheck( hipsparseDcsrilu02_bufferSize(
                        cusparseHandle, m, nnz, matLU, d_M_values,
                        d_A_rows, d_A_columns, infoM, &bufferSizeLU) );
    CUDACheck( hipMalloc(&d_bufferLU, bufferSizeLU) );
    CUSPARSECheck( hipsparseDcsrilu02_analysis(
                        cusparseHandle, m, nnz, matLU, d_M_values,
                        d_A_rows, d_A_columns, infoM,
                        HIPSPARSE_SOLVE_POLICY_USE_LEVEL, d_bufferLU) );
    int structural_zero;
    CUSPARSECheck( hipsparseXcsrilu02_zeroPivot(cusparseHandle, infoM,
                                                &structural_zero) );
    // M = L * U
    CUSPARSECheck( hipsparseDcsrilu02(
                        cusparseHandle, m, nnz, matLU, d_M_values,
                        d_A_rows, d_A_columns, infoM,
                        HIPSPARSE_SOLVE_POLICY_USE_LEVEL, d_bufferLU) );
    // Find numerical zero
    int numerical_zero;
    CUSPARSECheck( hipsparseXcsrilu02_zeroPivot(cusparseHandle, infoM,
                                                &numerical_zero) );

    CUSPARSECheck( hipsparseDestroyCsrilu02Info(infoM) );
    CUSPARSECheck( hipsparseDestroyMatDescr(matLU) );
    CUDACheck( hipFree(d_bufferLU) );
    //--------------------------------------------------------------------------
    // ### Run BiCGStab computation ###
    printf("BiCGStab loop:\n");
    // gpu_BiCGStab(cublasHandle, cusparseHandle, m,
    //              matA, matM_lower, matM_upper,
    //              d_B, d_X, d_R0, d_R, d_P, d_P_aux, d_S, d_S_aux, d_V, d_T,
    //              d_tmp, d_bufferMV, maxIterations, tolerance);


    const double zero      = 0.0;
    const double one       = 1.0;
    const double minus_one = -1.0;
    //--------------------------------------------------------------------------
    // Create opaque data structures that holds analysis data between calls
    double              coeff_tmp;
    size_t              bufferSizeL, bufferSizeU;
    void*               d_bufferL, *d_bufferU;
    hipsparseSpSVDescr_t spsvDescrL, spsvDescrU;
    CUSPARSECheck( hipsparseSpSV_createDescr(&spsvDescrL) );
    CUSPARSECheck( hipsparseSpSV_bufferSize(
                        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &coeff_tmp, matM_lower, d_P.vec, d_tmp.vec, HIP_R_64F,
                        HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, &bufferSizeL) );
    CUDACheck( hipMalloc(&d_bufferL, bufferSizeL) );
    CUSPARSECheck( hipsparseSpSV_analysis(
                        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &coeff_tmp, matM_lower, d_P.vec, d_tmp.vec, HIP_R_64F,
                        HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, d_bufferL) );

    // Calculate UPPER buffersize
    CUSPARSECheck( hipsparseSpSV_createDescr(&spsvDescrU) );
    CUSPARSECheck( hipsparseSpSV_bufferSize(
                        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &coeff_tmp, matM_upper, d_tmp.vec, d_P_aux.vec,
                        HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU,
                        &bufferSizeU) );
    CUDACheck( hipMalloc(&d_bufferU, bufferSizeU) );
    CUSPARSECheck( hipsparseSpSV_analysis(
                        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &coeff_tmp, matM_upper, d_tmp.vec, d_P_aux.vec,
                        HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU,
                        d_bufferU) );
    //--------------------------------------------------------------------------
    // ### 1 ### R0 = b - A * X0 (using initial guess in X)
    //    (a) copy b in R0
    CUDACheck( hipMemcpy(d_R0.ptr, d_B.ptr, m * sizeof(double),
                           hipMemcpyDeviceToDevice) );
    //    (b) compute R = -A * X0 + R
    CUSPARSECheck( hipsparseSpMV(cusparseHandle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &minus_one, matA, d_X.vec, &one, d_R0.vec,
                                 HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT,
                                 d_bufferMV) );
    //--------------------------------------------------------------------------
    double alpha, delta, delta_prev, omega;
    CUBLASCheck( hipblasDdot(cublasHandle, m, d_R0.ptr, 1, d_R0.ptr, 1,
                             &delta) );
    delta_prev = delta;
    // R = R0
    CUDACheck( hipMemcpy(d_R.ptr, d_R0.ptr, m * sizeof(double),
                           hipMemcpyDeviceToDevice) );
    //--------------------------------------------------------------------------
    // nrm_R0 = ||R||
    double nrm_R;
    CUBLASCheck( hipblasDnrm2(cublasHandle, m, d_R0.ptr, 1, &nrm_R) );
    double threshold = tolerance * nrm_R;
    printf("  Initial Residual: Norm %e' threshold %e\n", nrm_R, threshold);
    //--------------------------------------------------------------------------
    // ### 2 ### repeat until convergence based on max iterations and
    //           and relative residual
    for (int i = 1; i <= limit; i++) {
        // printf("  Iteration = %d; Error Norm = %e\n", i, nrm_R);
        //----------------------------------------------------------------------
        // ### 4, 7 ### P_i = R_i
        CUDACheck( hipMemcpy(d_P.ptr, d_R.ptr, m * sizeof(double),
                               hipMemcpyDeviceToDevice) );
        if (i > 1) {
            //------------------------------------------------------------------
            // ### 6 ### beta = (delta_i / delta_i-1) * (alpha / omega_i-1)
            //    (a) delta_i = (R'_0, R_i-1)
            CUBLASCheck( hipblasDdot(cublasHandle, m, d_R0.ptr, 1, d_R.ptr, 1,
                                     &delta) );
            //    (b) beta = (delta_i / delta_i-1) * (alpha / omega_i-1);
            double beta = (delta / delta_prev) * (alpha / omega);
            delta_prev  = delta;
            //------------------------------------------------------------------
            // ### 7 ### P = R + beta * (P - omega * V)
            //    (a) P = - omega * V + P
            double minus_omega = -omega;
            CUBLASCheck( hipblasDaxpy(cublasHandle, m, &minus_omega, d_V.ptr, 1,
                                      d_P.ptr, 1) );
            //    (b) P = beta * P
            CUBLASCheck( hipblasDscal(cublasHandle, m, &beta, d_P.ptr, 1) );
            //    (c) P = R + P
            CUBLASCheck( hipblasDaxpy(cublasHandle, m, &one, d_R.ptr, 1,
                                      d_P.ptr, 1) );
        }
        //----------------------------------------------------------------------
        // ### 9 ### P_aux = M_U^-1 M_L^-1 P_i
        //    (a) M_L^-1 P_i => tmp    (triangular solver)
        CUDACheck( hipMemset(d_tmp.ptr,   0x0, m * sizeof(double)) );
        CUDACheck( hipMemset(d_P_aux.ptr, 0x0, m * sizeof(double)) );
        CUSPARSECheck( hipsparseSpSV_solve(cusparseHandle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &one, matM_lower, d_P.vec, d_tmp.vec,
                                           HIP_R_64F,
                                           HIPSPARSE_SPSV_ALG_DEFAULT,
                                           spsvDescrL) );
        //    (b) M_U^-1 tmp => P_aux    (triangular solver)
        CUSPARSECheck( hipsparseSpSV_solve(cusparseHandle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &one, matM_upper, d_tmp.vec,
                                           d_P_aux.vec, HIP_R_64F,
                                           HIPSPARSE_SPSV_ALG_DEFAULT,
                                           spsvDescrU) );
        //----------------------------------------------------------------------
        // ### 10 ### alpha = (R'0, R_i-1) / (R'0, A * P_aux)
        //    (a) V = A * P_aux
        CUSPARSECheck( hipsparseSpMV(cusparseHandle,
                                     HIPSPARSE_OPERATION_NON_TRANSPOSE, &one,
                                     matA, d_P_aux.vec, &zero, d_V.vec,
                                     HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT,
                                     d_bufferMV) );
        //    (b) denominator = R'0 * V
        double denominator;
        CUBLASCheck( hipblasDdot(cublasHandle, m, d_R0.ptr, 1, d_V.ptr, 1,
                                 &denominator) );
        alpha = delta / denominator;
        // PRINT_INFO(delta)
        // PRINT_INFO(alpha)
        //----------------------------------------------------------------------
        // ### 11 ###  X_i = X_i-1 + alpha * P_aux
        CUBLASCheck( hipblasDaxpy(cublasHandle, m, &alpha, d_P_aux.ptr, 1,
                                  d_X.ptr, 1) );
        //----------------------------------------------------------------------
        // ### 12 ###  S = R_i-1 - alpha * (A * P_aux)
        //    (a) S = R_i-1
        CUDACheck( hipMemcpy(d_S.ptr, d_R.ptr, m * sizeof(double),
                               hipMemcpyDeviceToDevice) );
        //    (b) S = -alpha * V + R_i-1
        double minus_alpha = -alpha;
        CUBLASCheck( hipblasDaxpy(cublasHandle, m, &minus_alpha, d_V.ptr, 1,
                                  d_S.ptr, 1) );
        //----------------------------------------------------------------------
        // ### 13 ###  check ||S|| < threshold
        double nrm_S;
        CUBLASCheck( hipblasDnrm2(cublasHandle, m, d_S.ptr, 1, &nrm_S) );
        // PRINT_INFO(nrm_S)
        iter++;
        if (nrm_S < threshold)
            break;
        //----------------------------------------------------------------------
        // ### 14 ### S_aux = M_U^-1 M_L^-1 S
        //    (a) M_L^-1 S => tmp    (triangular solver)
        hipMemset(d_tmp.ptr, 0x0, m * sizeof(double));
        hipMemset(d_S_aux.ptr, 0x0, m * sizeof(double));
        CUSPARSECheck( hipsparseSpSV_solve(cusparseHandle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &one, matM_lower, d_S.vec, d_tmp.vec,
                                           HIP_R_64F,
                                           HIPSPARSE_SPSV_ALG_DEFAULT,
                                           spsvDescrL) );
        //    (b) M_U^-1 tmp => S_aux    (triangular solver)
        CUSPARSECheck( hipsparseSpSV_solve(cusparseHandle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &one, matM_upper, d_tmp.vec,
                                           d_S_aux.vec, HIP_R_64F,
                                           HIPSPARSE_SPSV_ALG_DEFAULT,
                                           spsvDescrU) );
        //----------------------------------------------------------------------
        // ### 15 ### omega = (A * S_aux, s) / (A * S_aux, A * S_aux)
        //    (a) T = A * S_aux
        CUSPARSECheck( hipsparseSpMV(cusparseHandle,
                                     HIPSPARSE_OPERATION_NON_TRANSPOSE, &one,
                                     matA, d_S_aux.vec, &zero, d_T.vec,
                                     HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT,
                                     d_bufferMV) );
        //    (b) omega_num = (A * S_aux, s)
        double omega_num, omega_den;
        CUBLASCheck( hipblasDdot(cublasHandle, m, d_T.ptr, 1, d_S.ptr, 1,
                                 &omega_num) );
        //    (c) omega_den = (A * S_aux, A * S_aux)
        CUBLASCheck( hipblasDdot(cublasHandle, m, d_T.ptr, 1, d_T.ptr, 1,
                                 &omega_den) );
        //    (d) omega = omega_num / omega_den
        omega = omega_num / omega_den;
        // PRINT_INFO(omega)
        // ---------------------------------------------------------------------
        // ### 16 ### omega = X_i = X_i-1 + alpha * P_aux + omega * S_aux
        //    (a) X_i has been updated with h = X_i-1 + alpha * P_aux
        //        X_i = omega * S_aux + X_i
        CUBLASCheck( hipblasDaxpy(cublasHandle, m, &omega, d_S_aux.ptr, 1,
                                  d_X.ptr, 1) );
        // ---------------------------------------------------------------------
        // ### 17 ###  R_i+1 = S - omega * (A * S_aux)
        //    (a) copy S in R
        CUDACheck( hipMemcpy(d_R.ptr, d_S.ptr, m * sizeof(double),
                               hipMemcpyDeviceToDevice) );
        //    (a) R_i+1 = -omega * T + R
        double minus_omega = -omega;
        CUBLASCheck( hipblasDaxpy(cublasHandle, m, &minus_omega, d_T.ptr, 1,
                                  d_R.ptr, 1) );
       // ---------------------------------------------------------------------
        // ### 18 ###  check ||R_i|| < threshold
        CUBLASCheck( hipblasDnrm2(cublasHandle, m, d_R.ptr, 1, &nrm_R) );
        // PRINT_INFO(nrm_R)
        if (nrm_R < threshold)
            break;
    }
    //--------------------------------------------------------------------------
    printf("Check Solution\n"); // ||R = b - A * X||
    //    (a) copy b in R
    CUDACheck( hipMemcpy(d_R.ptr, d_B.ptr, m * sizeof(double),
                           hipMemcpyDeviceToDevice) );
    // R = -A * X + R
    CUSPARSECheck( hipsparseSpMV(cusparseHandle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE, &minus_one,
                                 matA, d_X.vec, &one, d_R.vec, HIP_R_64F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, d_bufferMV) );
    // check ||R||
    CUBLASCheck( hipblasDnrm2(cublasHandle, m, d_R.ptr, 1, &nrm_R) );
    CUDACheck( hipMemcpy(h_X, d_X.ptr, m * sizeof(double),
                           hipMemcpyDeviceToHost) );
    std::vector<Scalar> xx(m);
    for (int i = 0; i < m; i++)
    {
        xx[i] = h_X[i];
    }
    
    printf("Final iterations: %d error norm = %e\n", iter, nrm_R);
    //--------------------------------------------------------------------------
    CUSPARSECheck( hipsparseSpSV_destroyDescr(spsvDescrL) );
    CUSPARSECheck( hipsparseSpSV_destroyDescr(spsvDescrU) );
    CUDACheck( hipFree(d_bufferL) );
    CUDACheck( hipFree(d_bufferU) );

    x.setvalues({xx.begin(), xx.end()});

    //--------------------------------------------------------------------------
    // ### Free resources ###
    CUSPARSECheck( hipsparseDestroyDnVec(d_B.vec) );
    CUSPARSECheck( hipsparseDestroyDnVec(d_X.vec) );
    CUSPARSECheck( hipsparseDestroyDnVec(d_R.vec) );
    CUSPARSECheck( hipsparseDestroyDnVec(d_R0.vec) );
    CUSPARSECheck( hipsparseDestroyDnVec(d_P.vec) );
    CUSPARSECheck( hipsparseDestroyDnVec(d_P_aux.vec) );
    CUSPARSECheck( hipsparseDestroyDnVec(d_S.vec) );
    CUSPARSECheck( hipsparseDestroyDnVec(d_S_aux.vec) );
    CUSPARSECheck( hipsparseDestroyDnVec(d_V.vec) );
    CUSPARSECheck( hipsparseDestroyDnVec(d_T.vec) );
    CUSPARSECheck( hipsparseDestroyDnVec(d_tmp.vec) );
    CUSPARSECheck( hipsparseDestroySpMat(matA) );
    CUSPARSECheck( hipsparseDestroySpMat(matM_lower) );
    CUSPARSECheck( hipsparseDestroySpMat(matM_upper) );
    CUSPARSECheck( hipsparseDestroy(cusparseHandle) );
    CUBLASCheck( hipblasDestroy(cublasHandle) );

    free(h_A_rows);
    free(h_A_columns);
    free(h_A_values);
    free(h_X);

    CUDACheck( hipFree(d_X.ptr) );
    CUDACheck( hipFree(d_B.ptr) );
    CUDACheck( hipFree(d_R.ptr) );
    CUDACheck( hipFree(d_R0.ptr) );
    CUDACheck( hipFree(d_P.ptr) );
    CUDACheck( hipFree(d_P_aux.ptr) );
    CUDACheck( hipFree(d_S.ptr) );
    CUDACheck( hipFree(d_S_aux.ptr) );
    CUDACheck( hipFree(d_V.ptr) );
    CUDACheck( hipFree(d_T.ptr) );
    CUDACheck( hipFree(d_tmp.ptr) );
    CUDACheck( hipFree(d_A_values) );
    CUDACheck( hipFree(d_A_columns) );
    CUDACheck( hipFree(d_A_rows) );
    CUDACheck( hipFree(d_M_values) );
    CUDACheck( hipFree(d_bufferMV) );
    return ;
}